#include "hip/hip_runtime.h"
#include <cstdlib> 
#include <cstdio>
#include <vector>
#include <iostream>
#include <pthread.h>
#include <chrono>
#include <string>

#include <opencv2/opencv.hpp>

#include "Scene.h"
#include "primitives/Obj.h"
#include "CSG.h"


using namespace std;
using namespace glm;
using namespace std::chrono;



string arg_to_string(char* a) { 
    int i; 
    string s = ""; 
    for (i = 0; i < 1000; i++) { 
    	if (a[i] == 0) break;
        s = s + a[i]; 
    } 
    return s;
} 


// Kernel function to add the elements of two arrays
__global__
void add(int n, vec3 *x, vec3 *y, float *z) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    vec3 curx, cury;

    for (int i = index; i < n; i += stride){
    	curx = x[i];
    	cury = y[i];

        z[i] = dot(curx,cury);
    }
}

void add_serial(int n, vec3 *x, vec3 *y, float *z) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    vec3 curx, cury;

    for (int i = index; i < n; i += stride){
    	curx = x[i];
    	cury = y[i];

        z[i] = dot(curx,cury);
    }
}



int main(void) {


	string filename = "/home/will/projects/cpprtx/meshes/torus.obj";

  	Mesh *torus = new Mesh(filename);
  	auto start = high_resolution_clock::now(); 

  	//return 0;

    int N = 1<<10;
    vec3 *x, *y;
    float *z;

    // Allocate Unified Memory – accessible from CPU or GPU
	//hipMallocManaged(&x, N*sizeof(vec3));
	//hipMallocManaged(&y, N*sizeof(vec3));
	//hipMallocManaged(&z, N*sizeof(float));

    x = malloc()

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
	    x[i] = vec3(1.0,0.0,0.0);
	    y[i] = vec3(1.0,0.0,0.0);
	}

	int blockSize = 8;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, blockSize>>>(N, x, y, z);
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	
	for (int i = 0; i < N; i++){ 
		maxError += z[i];
	}

	std::cout << "Max error: " << maxError << std::endl;
	
	// Free memory
	hipFree(x);
	hipFree(y);

	auto stop = high_resolution_clock::now(); 
	auto duration = duration_cast<milliseconds>(stop - start); 
	cout << "Elapsed: " << duration.count() << endl; 
	  
	return 0;
}