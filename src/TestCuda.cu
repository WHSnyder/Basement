#include "hip/hip_runtime.h"
#include <cstdlib> 
#include <cstdio>
#include <vector>
#include <iostream>
#include <pthread.h>
#include <chrono>
#include <string>

#include <opencv2/opencv.hpp> 

#include "Scene.h"
#include "primitives/Obj.h"
#include "CSG.h"

using namespace std;
using namespace glm;
using namespace std::chrono;



string arg_to_string(char* a) { 
    int i; 
    string s = ""; 
    for (i = 0; i < 1000; i++) { 
    	if (a[i] == 0) break;
        s = s + a[i]; 
    } 
    return s;
} 


// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}



int main(void) {

	auto start = high_resolution_clock::now(); 


	string filename = "/home/will/projects/cpprtx/meshes/torus.obj";

  	Mesh *torus = new Mesh(filename);

  	//return 0;

    int N = 1<<27;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
	    x[i] = 1.0f;
	    y[i] = 2.0f;
	}

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, blockSize>>>(N, x, y);
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	
	for (int i = 0; i < N; i++){ 
		maxError = fmax(maxError, fabs(y[i]-4.0f));
	    //std::cout << "Max error: " << maxError << std::endl;
	}

	// Free memory
	hipFree(x);
	hipFree(y);

	auto stop = high_resolution_clock::now(); 
	auto duration = duration_cast<milliseconds>(stop - start); 
	cout << "Elapsed: " << duration.count() << endl; 
	  
	return 0;
}