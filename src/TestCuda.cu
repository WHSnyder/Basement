#include "hip/hip_runtime.h"
#include <cstdlib> 
#include <cstdio>
#include <vector>
#include <iostream>
#include <pthread.h>
#include <chrono>
#include <regex> 
#include <string>

using namespace std;
using namespace std::chrono;

regex object_header("o ");
regex vertex_decl("v( [-]?[0-9]*\.?[0-9]*){3}");
regex float_decl("([-]?[0-9]*\.?[0-9]*)");

smatch sm;


void read_obj_file(string filename){

	ifstream file (filename); //file just has some sentences
	int i = 0;
	
	if (!file) {
		cout << "unable to open file";
		return false;
	}

	string line;

	while (getline (file, line)) {

		if (regex_search(line, sm, vertex_decl)){
			cout << "Vertex " << i++ << ": ";
			if (regex_search(line, sm, float_decl)) {
			    for (int i=1; i<sm.size(); i++) {
			        cout << sm[i] << " ";
			    }
			}			
			cout << endl;
		}
	}
} 




string arg_to_string(char* a) { 
    int i; 
    string s = ""; 
    for (i = 0; i < 1000; i++) { 
    	if (a[i] == 0) break;
        s = s + a[i]; 
    } 
    return s;
} 


// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++){   
        y[i] = x[i] + y[i];
    }
}



int main(void) {

  	string filename = "/home/will/projects/cpprtx/meshes/torus.obj"; // = arg_to_string(argv[1]);

  	read_obj_file(filename);

  	return 0;


    int N = 1<<27;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
	    x[i] = 1.0f;
	    y[i] = 2.0f;
	}

	// Run kernel on 1M elements on the GPU
	add<<<1, 1>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	
	for (int i = 0; i < N; i++){ 
		maxError = fmax(maxError, fabs(y[i]-4.0f));
	    std::cout << "Max error: " << maxError << std::endl;
	}

	// Free memory
	hipFree(x);
	hipFree(y);
	  
	return 0;
}